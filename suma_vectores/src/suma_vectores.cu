/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

//System includes
#include <stdio.h>
#include <stdlib.h>

//Cuda includes
#include <hip/hip_runtime.h>


__global__ void vecAddKernel(float *A, float *B, float *C, int n){
	int i = threadIdx.x+blockDim.x*blockIdx.x;
	if(i<n)
		C[i] = A[i]+B[i];
}

extern "C"
void vecAdd_par(float* h_A, float* h_B, float* h_C, int n){
	int size = n*sizeof(float);
	float *dA, *dB, *dC;

	// Alocate memory para A, B, C
	hipMalloc((void **) &dA,size);
	hipMalloc((void **) &dB,size);
	hipMalloc((void **) &dC,size);

	// Copiar A y B a la memoria del dispositivo
	hipMemcpy(dA,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,h_B,size,hipMemcpyHostToDevice);

	// llamada al kernel
	// dA, dB, dC tal y como lo hemos visto antes
	// Para bloques de hebras de 256
	int var=256;
	if(n>512)
		var=512;
	dim3 DimBlock(var,1,1);
	dim3 DimGrid(((n-1)/DimBlock.x)+1,1,1);
	vecAddKernel<<<DimGrid,DimBlock>>>(dA,dB,dC,n);

	// copiar C desde el dispositivo
	hipMemcpy(h_C,dC,size,hipMemcpyDeviceToHost);

	// liberar memoria de A, B y C
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}
